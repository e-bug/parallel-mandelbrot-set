
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>


/******************************************************************************/
#define BLOCK_SIZE_X 256
#define BLOCK_SIZE_Y 1

__constant__ int c_maxiter;
__constant__ double c_xmin;
__constant__ double c_ymin;
__constant__ double c_x_step;
__constant__ double c_y_step;
__constant__ int c_N;
__constant__ int c_width;
__constant__ int c_rowsize;

/******************************************************************************/

__device__ void d_smooth_fast_element_colormap(int iter, double re2, double im2,
                                               int *rp, int *gp, int *bp)
{
  if(iter == c_maxiter) {
    /* black */
    *rp = 0; // Red channel
    *gp = 0; // Green channel
    *bp = 0; // Blue channel
  }
  else {
    int brightness = 256.*log2(1.75-log2(0.5)+iter-log2(log2(re2+im2)))/log2((double)c_maxiter);

    *rp = brightness; // Red channel
    *gp = brightness; // Green channel
    *bp = 255; // Blue channel
  }
}


__device__ void in_cardioid_or_period2_bulb(int *iterp, double x, double y)
{
  double xdiff = x - 0.25;
  double y2 = y * y;
  double q = xdiff*xdiff + y2;

  // Is the point in the cardioid?
  if (q * (q + xdiff) < 0.25*y2) {
    *iterp = c_maxiter;
  }
  else if ((x+1.)*(x+1.) + y2 < 0.0625) { // Is the point in the period-2 bulb?
    *iterp = c_maxiter;
  }

}


__global__ void compute_escape_time(char *img)
{
  int offset = blockIdx.x*blockDim.x + threadIdx.x + (blockIdx.y*blockDim.y + threadIdx.y) * (gridDim.x * blockDim.x);
  int i = offset / c_width;
  int j = offset - i * c_width;
  int iteration = 0;
  double c_re = c_xmin + c_x_step/2 + j*c_x_step;
  double c_im = c_ymin + c_y_step/2 + i*c_y_step;
  double zn_re = 0.;
  double zn_im = 0.;
  double tmp_re;
  double re2 = 0.;
  double im2 = 0.;
  int bailout_radius2 = 2*2;
  int r, g, b;

  if (offset < c_N) {
    // Check if point is in cardioid or in period-2 bulb
    in_cardioid_or_period2_bulb(&iteration, c_re, c_im);

    while ((re2 + im2 < bailout_radius2) && (iteration < c_maxiter)) {
      tmp_re = re2 - im2 + c_re;
      zn_im = zn_re * zn_im;
      zn_im += zn_im; // Multiply by two
      zn_im += c_im;
      zn_re = tmp_re;

      re2 = zn_re * zn_re;
      im2 = zn_im * zn_im;
      iteration++;
    }

    d_smooth_fast_element_colormap(iteration, re2, im2, &r, &g, &b);

    offset = c_rowsize * i + 3 * j; // offset in the image array
    img[offset++] = b;
    img[offset++] = g;
    img[offset] = r;
  }
}

/******************************************************************************/

extern "C" void kernel_wrapper(char *h_img, int d_img_size, int MAX_ITER,
                               double X_MIN, double Y_MIN, double h_x_step,
                              double h_y_step, int N, int WIDTH, int row_size)
{

  dim3 block_size, grid_size;
  char *d_img;

  // Create the grid of blocks of threads
  block_size.x = BLOCK_SIZE_X; block_size.y = BLOCK_SIZE_Y;
  grid_size.x = N / (block_size.x*block_size.y) + (N%(block_size.x*block_size.y) == 0? 0 : 1);

  hipMalloc((void **)&d_img, d_img_size);
  hipMemset(d_img, 0, d_img_size);

  // Copy memory to constant memory in the device
  hipMemcpyToSymbol(HIP_SYMBOL(c_maxiter), &MAX_ITER, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(c_xmin), &X_MIN, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(c_ymin), &Y_MIN, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(c_x_step), &h_x_step, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(c_y_step), &h_y_step, sizeof(double));
  hipMemcpyToSymbol(HIP_SYMBOL(c_N), &N, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(c_width), &WIDTH, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(c_rowsize), &row_size, sizeof(int));

  // Call the kernel to execute on the gpu
  compute_escape_time<<<grid_size, block_size>>>(d_img);

  // Copy the results back
  hipMemcpy(h_img, d_img, d_img_size, hipMemcpyDeviceToHost);

  hipFree(d_img);
}
